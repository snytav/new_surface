#include "hip/hip_runtime.h"

#include "surf2Dread.h"
#include <stdio.h>
#include <iostream>
#include "cuPrintf.cuh"

SurfaceEmulator *d_surf;

__global__ void kernel(SurfaceEmulator **d_s,int n,int m){
	*d_s = new SurfaceEmulator(m,n);
	(*d_s)->write(2,3,3.14);
	double d = (*d_s)->read(2,3);
 	cuPrintf("res  \n");

}


int main(){

  cudaPrintfInit();	
  kernel<<<1,1>>>(&d_surf,3,5);
  hipDeviceSynchronize();
  cudaPrintfDisplay (stdout, true);
  cudaPrintfEnd ();



}
