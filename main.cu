#include "hip/hip_runtime.h"

#include "surf2Dread.h"
#include <stdio.h>

SurfaceEmulator *d_surf;

__global__ void kernel(SurfaceEmulator **d_s,int n,int m){
	*d_s = new SurfaceEmulator(m,n);
	(*d_s)->write(2,3,3.14);
	double d = (*d_s)->read(2,3);
	printf("res %e \n",d);

}


int main(){

  kernel<<<1,1>>>(&d_surf,3,5);
  hipDeviceSynchronize();
}
