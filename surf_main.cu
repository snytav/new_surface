#include "hip/hip_runtime.h"
i///// https://forums.developer.nvidia.com/t/can-i-use-operator-new-in-device-code/37594ii

#include "surf2Dread.h"

SurfaceEmulater d_surf;

__global__ void kernel(SurfaceEmulator **d_s,in n,int m){
	*d_s = new SurfaceEmulator(m,n);

}


int main(){

  kernel<<<1,1>>>();
  hipDeviceSynchronize();
}
