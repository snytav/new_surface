#include "hip/hip_runtime.h"

#include "surf2Dread.h"

SurfaceEmulator *d_surf;

__global__ void kernel(SurfaceEmulator **d_s,int n,int m){
	*d_s = new SurfaceEmulator(m,n);

}


int main(){

  kernel<<<1,1>>>(&d_surf,3,5);
  hipDeviceSynchronize();
}
